
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>
#include <iomanip>
// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
    for (int i = 0; i < n; i++)
        y[i] = x[i] + y[i];
}

int main(void)
{
    int N = 1<<24;
    float *x, *y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    std::cout << std::fixed << std::setprecision(5);
    std::chrono::time_point<std::chrono::high_resolution_clock> start_time = std::chrono::high_resolution_clock::now();

    // Run kernel on 1M elements on the GPU
    add<<<1, 1>>>(N, x, y);

    std::chrono::time_point<std::chrono::high_resolution_clock> end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end_time - start_time;
    std::cout << " Elapsed time is : " << elapsed.count() << " " << std::endl;

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}
